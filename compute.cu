#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <stdio.h>

__global__ void ComputeKernel(vector3** accels,vector3 *hPos, double *mass){
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	//for (i=0;i<NUMENTITIES;i++){
	//	for (j=0;j<NUMENTITIES;j++){
	//cuPrintf("Curr Row: %d, Curr Col: %d",i,j);
	if(i<NUMENTITIES && j<NUMENTITIES){
		if (i==j) {
				FILL_VECTOR(accels[i][j],0,0,0);
			}
			else{
				vector3 distance;
				for (int k=0;k<3;k++) distance[k]=hPos[i][k]-hPos[j][k];
				double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
				double magnitude=sqrt(magnitude_sq);
				double accelmag=-1*GRAV_CONSTANT*mass[j]/magnitude_sq;
				FILL_VECTOR(accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
			}
	}
	//	}
	//}
}

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){
	//make an acceleration matrix which is NUMENTITIES squared in size;
	int i,j,k;
	vector3* values=(vector3*)malloc(sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	vector3** accels=(vector3**)malloc(sizeof(vector3*)*NUMENTITIES);
	//vector3* d_values;
	vector3** d_accels;
	//hipMalloc(&d_values, sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	hipMalloc(&d_accels, sizeof(vector3*)*NUMENTITIES);

	for (i=0;i<NUMENTITIES;i++){
		accels[i]=&values[i*NUMENTITIES];
		//hipMalloc(&accels[i], sizeof(vector3)*NUMENTITIES);
		//hipMemcpy(accels[i], &values[i*NUMENTITIES], sizeof(vector3)*NUMENTITIES, hipMemcpyHostToDevice);
	}
	hipMemcpy(d_accels, accels, sizeof(vector3*)*NUMENTITIES, hipMemcpyHostToDevice);
	//hipMemcpy(d_values, values, NUMENTITIES*NUMENTITIES*sizeof(vector3), hipMemcpyHostToDevice);
	//hipMemcpy(d_accels, accels, sizeof(vector3*)*NUMENTITIES, hipMemcpyDeviceToHost);
	printf("Last Error: %d",hipGetLastError());

	dim3 dimBlock(16, 16);
	dim3 dimGrid((NUMENTITIES / dimBlock.x)+1, (NUMENTITIES / dimBlock.y)+1);
	
	ComputeKernel<<<dimGrid, dimBlock>>>(d_accels,hPos,mass);
	//first compute the pairwise accelerations.  Effect is on the first argument.
	
	hipMemcpy(accels, d_accels, sizeof(vector3*)*NUMENTITIES, hipMemcpyDeviceToHost);
	//printf("Test: %f", accels[10][20][0]);

	/*
	hipMemcpy(accels, d_accels, sizeof(vector3*)*NUMENTITIES, hipMemcpyDeviceToHost);
	for(int l = 0; l< NUMENTITIES; l++){
		hipMemcpy(values + l, accels[l], sizeof(vector3), hipMemcpyDeviceToHost);
	}
	*/
	//printf("accles Size: %lu, d_accels Size: %lu, Correct Size: %lu",sizeof(accels),sizeof(d_accels),sizeof(vector3*)*NUMENTITIES);
	
	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	for (i=0;i<NUMENTITIES;i++){
		vector3 accel_sum={0,0,0};
		for (j=0;j<NUMENTITIES;j++){
			for (k=0;k<3;k++)
				accel_sum[k]+=accels[i][j][k];
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (k=0;k<3;k++){
			hVel[i][k]+=accel_sum[k]*INTERVAL;
			hPos[i][k]=hVel[i][k]*INTERVAL;
		}
	}

	free(accels);
	free(values);
	//hipFree(d_values);
	hipFree(d_accels);
}
